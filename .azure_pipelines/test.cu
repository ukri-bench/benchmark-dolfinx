#include <hip/hip_runtime.h>
#include <iostream>
int main()
{
  int devCount;
  hipError_t status = hipGetDeviceCount(&devCount);
  if (status != hipSuccess)
    throw std::runtime_error("Error getting device properties");
  std::cout << "Num devices: " << devCount << std::endl;

  hipDeviceProp_t props;
  status = hipGetDeviceProperties(&props, 0);
  if (status != hipSuccess)
    throw std::runtime_error("Error getting device properties");
  std::cout << "Device: " << props.name << ": " << props.major << "." << props.minor
            << std::endl;
}
